#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void hello_cuda(){
    printf("Hello CUDA world \n");
}

__global__ void hello_cuda_with_id(){
    printf("Hello CUDA world with blockIDx.x %d, threadIDx %d \n", blockIdx.x, threadIdx.x);
}

int main(){

    dim3 grid(2);
    dim3 block(4);
    hello_cuda_with_id<<<grid, block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;

}