#include <iostream>
#include <hip/hip_runtime.h>


__global__ void addOneKernel(int* d_data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_data[idx] += 1;
    }
}


__global__ void multiplyByTwoKernel(int* d_data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_data[idx] *= 2;
    }
}

void test_Stream()
{
    const int arraySize = 1<<18;
    const int byteSize = arraySize * sizeof(int);

    // Allocate memory for host and device
    int* d1_data, *d2_data;
    int* h1_data, *h2_data;
    hipMalloc(&d1_data, byteSize);
    hipMalloc(&d2_data, byteSize);
    hipHostMalloc(&h1_data, byteSize, hipHostMallocDefault);
    hipHostMalloc(&h2_data, byteSize, hipHostMallocDefault);
    // Initial
    for (int i = 0; i < arraySize; ++i) {
        h1_data[i] = i;
        h2_data[i] = i;
    }

    // Construct 2 cuda stream
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    // Set up grid, block
    int blockSize = 32;
    int gridSize = (arraySize + blockSize - 1) / blockSize;
    // Async from host to device
    hipMemcpyAsync(d1_data, h1_data, byteSize, hipMemcpyHostToDevice, stream1);
    // kernel function1 with stream1
    addOneKernel<<<gridSize, blockSize, 0, stream1>>>(d1_data, arraySize);
    hipMemcpyAsync(h1_data, d1_data, byteSize, hipMemcpyDeviceToHost, stream1);
    // kernel function2 with stream2
    hipMemcpyAsync(d2_data, h2_data, byteSize, hipMemcpyHostToDevice, stream2);
    addOneKernel<<<gridSize, blockSize, 0, stream2>>>(d2_data, arraySize);
    hipMemcpyAsync(h2_data, d2_data, byteSize, hipMemcpyDeviceToHost, stream2);
    // Wait
    hipStreamSynchronize(stream1);
    hipStreamDestroy(stream1); 
    hipStreamSynchronize(stream2);
    hipStreamDestroy(stream2);
    // Free
    hipFree(d1_data);
    hipFree(d2_data);
    hipHostFree(h1_data);
    hipHostFree(h2_data);
}

void test_Stream_With_Events()
{
    const int arraySize = 1<<18;
    const int byteSize = arraySize * sizeof(int);

    // Allocate memory for host and device
    int* d1_data, *d2_data, *d3_data;
    int* h1_data, *h2_data, *h3_data;
    hipMalloc(&d1_data, byteSize);
    hipMalloc(&d2_data, byteSize);
    hipMalloc(&d3_data, byteSize);
    hipHostMalloc(&h1_data, byteSize, hipHostMallocDefault);
    hipHostMalloc(&h2_data, byteSize, hipHostMallocDefault);
    hipHostMalloc(&h3_data, byteSize, hipHostMallocDefault);

    // Initial
    for (int i = 0; i < arraySize; ++i) {
        h1_data[i] = i;
        h2_data[i] = i;
        h3_data[i] = i;
    }

    // Construct Stream
    hipStream_t stream[3];
    for(int i = 0; i < 3; i++)
    {
        hipStreamCreate(&stream[i]);
    }
    
    // Construct Event
    hipEvent_t event;
    hipEventCreateWithFlags(&event, hipEventDisableTiming);

    // Set kernel config
    int blockSize = 32;
    int gridSize = (arraySize + blockSize - 1) / blockSize;
    hipMemcpyAsync(d1_data, h1_data, byteSize, hipMemcpyHostToDevice, stream[0]);
    addOneKernel<<<gridSize, blockSize, 0, stream[0]>>>(d1_data, arraySize);
    // Record and set wait
    hipEventRecord(event, stream[0]);
    hipStreamWaitEvent(stream[2],event,0);
    hipMemcpyAsync(h1_data, d1_data, byteSize, hipMemcpyDeviceToHost, stream[0]);
    // Second Stream
    hipMemcpyAsync(d2_data, h2_data, byteSize, hipMemcpyHostToDevice, stream[1]);
    addOneKernel<<<gridSize, blockSize, 0, stream[1]>>>(d2_data, arraySize);
    hipMemcpyAsync(h2_data, d2_data, byteSize, hipMemcpyDeviceToHost, stream[1]);
    // Third Stream
    hipMemcpyAsync(d3_data, h3_data, byteSize, hipMemcpyHostToDevice, stream[2]);
    addOneKernel<<<gridSize, blockSize, 0, stream[2]>>>(d3_data, arraySize);
    hipMemcpyAsync(h3_data, d3_data, byteSize, hipMemcpyDeviceToHost, stream[2]);

    // Synchronize and destroy stream
    for(int i = 0; i < 3; i++)
    {
        hipStreamSynchronize(stream[i]);
        hipStreamDestroy(stream[i]); 
    }

    // Destroy Event
    hipEventDestroy(event);
    
    // Free
    hipFree(d1_data);
    hipFree(d2_data);
    hipFree(d3_data);
    hipHostFree(h1_data);
    hipHostFree(h2_data);
    hipHostFree(h3_data);
}

int main() {


    test_Stream_With_Events();
    return 0;
}
